#include "hip/hip_runtime.h"
#include "cuda_test.h"

 unsigned char *X, *Y, *Z, *W;
 unsigned char *d_X, *d_Y, *d_Z, *d_W;
 unsigned char *d_offset;

void CPUMemAlloc()
{
  X = (unsigned char*)malloc(MAT_NUMEL * sizeof(unsigned char));
  Y = (unsigned char*)malloc(TOTAL_NUMEL * sizeof(unsigned char));
  Z = (unsigned char*)malloc(TOTAL_NUMEL * sizeof(unsigned char));
  W = (unsigned char*)malloc(TOTAL_NUMEL * sizeof(unsigned char));
}

void GPUMemAlloc()
{
  size_t size = MAT_NUMEL * sizeof(unsigned char);
  hipError_t msgErr = hipMalloc(&d_X, size);
  printf("Memory Allocate d_X: %s\n",hipGetErrorString(msgErr));

  size = TOTAL_NUMEL * sizeof(unsigned char);
  msgErr = hipMalloc(&d_Y, size);
  printf("Memory Allocate d_Y: %s\n",hipGetErrorString(msgErr));

  size = TOTAL_NUMEL * sizeof(unsigned char);
  msgErr = hipMalloc(&d_Z, size);
  printf("Memory Allocate d_Z: %s\n",hipGetErrorString(msgErr));

  size = TOTAL_NUMEL * sizeof(unsigned char);
  msgErr = hipMalloc(&d_W, size);
  printf("Memory Allocate d_W: %s\n",hipGetErrorString(msgErr));
}

void dataGenerating()
{
// (1) look-up table
  for(int row = 0; row < MAT_SIZE; row++)
   for(int col = 0; col < MAT_SIZE; col++)
    X[row * MAT_SIZE + col] = 1;

  for(int depth = 0; depth < NUM_MATR; depth++)
   for(int row = 0; row < MAT_SIZE; row++)
    for(int col = 0; col < MAT_SIZE; col++)
     Y[depth * MAT_NUMEL + row * MAT_SIZE + col] = depth;

//(2) assign random values to 256 matrices 100x100
  for(int depth = 0; depth < NUM_MATR; depth++)  
   for(int row = 0; row < MAT_SIZE; row++)
    for(int col = 0; col < MAT_SIZE; col++)
    {
      int index = depth * MAT_NUMEL + row * MAT_SIZE + col;
      Z[index] = rand() % 233;//256
      W[index] = Z[index];
    }
  printf("Matrix W: ");
  for(int i = 0; i < 5; i++) printf("%d ", W[i]);
  printf("... ");
  for(int i = 0; i < 5; i++) printf("%d ", W[MAT_NUMEL * NUM_MATR - i -1]);
  printf("\n\n");
}

void copyLookupTabGPU()
{
  size_t size = MAT_NUMEL * sizeof(unsigned char);
  hipError_t msgErr = hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);
  printf("hipMemcpy d_X: %s\n",hipGetErrorString(msgErr));
  size = TOTAL_NUMEL * sizeof(unsigned char);
  msgErr = hipMemcpy(d_Y, Y, size, hipMemcpyHostToDevice);
  printf("hipMemcpy d_Y: %s\n",hipGetErrorString(msgErr));

  unsigned char *offset;
  offset = (unsigned char*)malloc(NUM_MATR*sizeof(unsigned char));
  hipMalloc(&d_offset, NUM_MATR*sizeof(unsigned char)); 
  for (int depth = 0; depth < NUM_MATR; depth++)
  {
    unsigned char maxval = 0;
    for(int row = 0; row < MAT_SIZE; row++)
     for(int col = 0; col < MAT_SIZE; col++)
     {
       int index = depth * MAT_NUMEL + row * MAT_SIZE + col;
       if(Z[index] > maxval) maxval = Z[index];
     }
    offset[depth] = 255- maxval;
   }
   msgErr = hipMemcpy(d_offset, offset, NUM_MATR*sizeof(unsigned char), hipMemcpyHostToDevice);
}

void copyDataToGPU()
{
  size_t size = TOTAL_NUMEL * sizeof(unsigned char);
  hipError_t msgErr = hipMemcpy(d_Z, Z, size, hipMemcpyHostToDevice);
}

void copyDataFromGPU()
{
  size_t size = TOTAL_NUMEL * sizeof(unsigned char);
  hipMemcpy(W, d_W, size, hipMemcpyDeviceToHost);
}

__global__ void GPU_First_Task(unsigned char *offset, unsigned char *X, unsigned char *Z, unsigned char *W) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int depth = index / (MAT_NUMEL);
  int X_index = index % MAT_NUMEL;
  W[index] = Z[index] + X[X_index] * offset[depth]; 
}

__global__ void GPU_Second_Task(unsigned char *offset, unsigned char *Y, unsigned char *Z, unsigned char *W) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int depth = index / (MAT_NUMEL);
  W[index] = Z[index] + offset[depth];
}

int main()
{
  clock_t start, end;
  double cpu_time_used;
  CPUMemAlloc();
  dataGenerating();
  GPUMemAlloc();
  copyLookupTabGPU();

//----Speed of Vector Product/Multiplication-------//
  printf("\nSpeed of Vector Product/Multiplication:\n");

  start = clock();
  for (int i = 0; i < NUM_LOOP; i++) copyDataToGPU();
  end = clock();
  cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
  printf("copyDataToGPU took %f seconds to execute \n", cpu_time_used);

  start = clock();
  for(int i = 0; i < NUM_LOOP; i++)
  {
     GPU_First_Task<<<MAT_SIZE*MAT_SIZE, NUM_MATR>>>(d_offset, d_X, d_Z, d_W);
     hipDeviceSynchronize();
  }
  end = clock();
  cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
  printf("Kernel took %f seconds to execute \n", cpu_time_used);

  start = clock();
  for (int i = 0; i < NUM_LOOP; i++) copyDataFromGPU();
  end = clock();
  cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
  printf("copyDataFromGPU took %f seconds to execute \n", cpu_time_used);

  printf("Matrix W: ");
  for(int i = 0; i < 5; i++) printf("%d ", W[i]);
  printf("... ");
  for(int i = 0; i < 5; i++) printf("%d ", W[MAT_NUMEL * NUM_MATR - i -1]);

//-----------------Speed of Addition----------------//
  printf("\n\nSpeed of Addition using look-up table:\n");
  start = clock();
  for (int i = 0; i < NUM_LOOP; i++) copyDataToGPU();
  end = clock();
  cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
  printf("copyDataToGPU took %f seconds to execute \n", cpu_time_used);

  start = clock();
  for(int i = 0; i < NUM_LOOP; i++)
  {
     GPU_Second_Task<<<MAT_SIZE*MAT_SIZE, NUM_MATR>>>(d_offset, d_Y, d_Z, d_W);
     hipDeviceSynchronize();
  }
  end = clock();
  cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
  printf("Kernel took %f seconds to execute \n", cpu_time_used);

  start = clock();
  for (int i = 0; i < NUM_LOOP; i++) copyDataFromGPU();
  end = clock();
  cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
  printf("copyDataFromGPU took %f seconds to execute \n", cpu_time_used);

  printf("Matrix W: ");
  for(int i = 0; i < 5; i++) printf("%d ", W[i]);
  printf("... ");
  for(int i = 0; i < 5; i++) printf("%d ", W[MAT_NUMEL * NUM_MATR - i -1]);
  printf("\n");

// Free device memory
  hipFree(d_X);
  hipFree(d_Y);
  hipFree(d_Z);
  hipFree(d_W);
  hipFree(d_offset);
// Free host memory
  free(X);
  free(Y);
  free(Z);
  free(W);

  return 0;
}


